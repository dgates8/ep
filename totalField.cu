
#include <hip/hip_runtime.h>
#include <fstream>
#include <iterator>
#include <vector>
#include <iostream>
#include <cstdlib>
#include <string>
#include <sstream>
#include <iomanip>
#include <math.h>
#include <stdio.h>

#define energySize 16000000
#define blockSize 256

//define macro for error checking
#define cudaCheckError(){	   												  \
	hipError_t err = hipGetLastError();											  \
	if(err != hipSuccess){              											  \
		std::cout << "Error in " << __FILE__ << " at line " << __LINE__ << " : " <<  hipGetErrorString(err) << std::endl; \
		exit(EXIT_FAILURE);                 										  \
	}                                     											  \
}

void getSourceFile(std::vector<double>& eNomVec, std::vector<double>& rangeVec, 
			std::vector<double>& sigmaXVec,std::vector<double>& sigmaYVec, 
			std::vector<double>& eMeanVec, std::vector<double>& sigmaEVec, 
			std::vector<double>& xVec, std::vector<double>& yVec, 
			std::vector<double>& nxVec,std::vector<double>& nyVec,
			std::vector<double>& weightVec, int& numGroups) 
{
	int dateOfMeasurement;
	long int numberOfGroups;
	double  eNom, range, sigmaX, sigmaY, eMean, sigmaE, xcoord, ycoord, weight, nx, ny;
	
	std::string line;
	//declare and open file
	std::ifstream ifile("IMPT_source.dat", std::ios::in);
	if(!ifile){
		std::cout << "Error, IMPT_source not found" << std::endl;
	}else{
		//read in date of measurement
		ifile >> dateOfMeasurement;
	
		//read in number of groups
		ifile >> numberOfGroups;
		numGroups = numberOfGroups;
		
		//skip over header line
		std::string e, r, x, y, m, s, nx1, ny1, x1, y1, w;
		ifile >> e;
		ifile >> r;
		ifile >> x;
		ifile >> y;
		ifile >> m;
		ifile >> s;
		ifile >> x1;
		ifile >> y1;
		ifile >> nx1;
		ifile >> ny1;
		ifile >> w;
	
		//intialize memory for faster read in 
//		xVec.reserve(numberOfGroups);
//		yVec.reserve(numberOfGroups);
//		nxVec.reserve(numberOfGroups);
//		nyVec.reserve(numberOfGroups);
		weightVec.reserve(numberOfGroups);
		eNomVec.reserve(numberOfGroups);					
		//read in data to vectors
		for(int i = 0; i < numberOfGroups; i++){
			ifile >> eNom;
			ifile >> range;
			ifile >> sigmaX;
			ifile >> sigmaY;
			ifile >> eMean;
			ifile >> sigmaE;
			ifile >> xcoord;
			ifile >> ycoord;
			ifile >> nx;
			ifile >> ny;
			ifile >> weight;

			eNomVec.push_back(eNom);
//			rangeVec.push_back(range);
//			sigmaXVec.push_back(sigmaX);
//			sigmaYVec.push_back(sigmaY);
//			eMeanVec.push_back(eMean);
//			xVec.push_back(xcoord);
//			yVec.push_back(ycoord);
//			nxVec.push_back(nx);
//			nyVec.push_back(ny);
			weightVec.push_back(weight);
		}
	}
}

int main(int argc, char** argv){
	int numberOfGroups;
	std::vector<double> eNom, range, sigmaX, sigmaY, eMean, sigmaE, xCoord, yCoord, nx, ny, weight;
	getSourceFile(eNom, range, sigmaX, sigmaY, eMean, sigmaE, xCoord, yCoord, nx, ny, weight, numberOfGroups);
	
	double* finalEnergy;
	finalEnergy = (double*)malloc(energySize*sizeof(double));
	memset(finalEnergy, 0, energySize*sizeof(double));	
	//do not use device 1 for anything
	hipSetDevice(2);

	for(int master = atoi(argv[1])-1; master < atoi(argv[2]); master++){
		
		//declare stream size variables and open file/check for errors
		std::streampos bufferSize;

		//create fileName to read in data
		std::ostringstream fName;
		if(master < 9){
			fName << std::fixed << "PercentEdep3D_field_0" << master+1 << "_" << std::setprecision(1) << eNom[master] << "MeV.bin";
		}else{
			fName << std::fixed << "PercentEdep3D_field_" << master+1 << "_"  << std::setprecision(1) << eNom[master] << "MeV.bin";

		}
		std::string fileName = fName.str();
		std::cout << fileName << std::endl;
		std::ifstream ifile(fileName.c_str(), std::ios::in | std::ios::binary);
		if(!ifile){
			std::cout << "Error, no file found" << std::endl;
			exit(1);
		}
		
		//get file size
		ifile.seekg(0, std::ios::end);
		bufferSize = ifile.tellg();
		ifile.seekg(0, std::ios::beg);

		//declare buffer
		std::vector<double> buffer(bufferSize/sizeof(double));
		
		//read in data
		ifile.read(reinterpret_cast<char*>(buffer.data()), bufferSize); 

		int size = bufferSize/(sizeof(double)*400);
		
		//copy memory from buffer to energy
		double *energy;
		energy = (double*)malloc(size*sizeof(double)*400);
		std::copy(buffer.begin(), buffer.end(), energy);

		for(int i = 0; i < energySize ;i++){
			finalEnergy[i] += energy[i];
		}
	}//end of master loop
	
	std::cout << "Writing to binary file" << std::endl;
	std::ostringstream OName;
	OName << std::fixed << "PercentEdep3D_field_" << argv[1] << "-" << argv[2] << ".bin";
	std::string fileNameOut = OName.str();
	std::ofstream ofile(fileNameOut.c_str() , std::ios::out | std::ios::binary);
	ofile.write(reinterpret_cast<char*>(finalEnergy), energySize*sizeof(double));

}
